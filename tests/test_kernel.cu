
#include <hip/hip_runtime.h>
#include <cstdio>

// Declare the function from the CUDA module
extern "C" void call_hello_kernel_();

int main() {
    printf("Running CUDA kernel test...\n");
    call_hello_kernel_();
    printf("CUDA kernel test completed.\n");
    return 0;
}
